#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <hip/hip_runtime.h>

namespace py = pybind11;

// Add these declarations for the kernels
extern __global__ void vectorAddKernel(float* A, float* B, float* C, int numElements);
extern __global__ void reluKernel(float* input, float* output, int numElements);

void vectorAddCUDA(float* d_A, float* d_B, float* d_C, int numElements, int threadsPerBlock) {
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipDeviceSynchronize();  // Ensure kernel finishes
}

void reluCUDA(float* d_input, float* d_output, int numElements, int threadsPerBlock) {
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    reluKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, numElements);
    hipDeviceSynchronize();  // Ensure kernel finishes
}

PYBIND11_MODULE(template, m) {
    m.def("vector_add", [](py::array_t<float> A, py::array_t<float> B) {
        if (A.size() != B.size()) throw std::runtime_error("Arrays must be the same size");
        size_t numElements = A.size();
        
        // Allocate device memory
        float *d_A, *d_B, *d_C;
        hipMalloc(&d_A, numElements * sizeof(float));
        hipMalloc(&d_B, numElements * sizeof(float));
        hipMalloc(&d_C, numElements * sizeof(float));
        
        // Copy data to device
        hipMemcpy(d_A, A.data(), numElements * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B.data(), numElements * sizeof(float), hipMemcpyHostToDevice);
        
        // Run kernel
        int threadsPerBlock = 256;  // Common choice
        vectorAddCUDA(d_A, d_B, d_C, numElements, threadsPerBlock);
        
        // Copy result back
        py::array_t<float> result(numElements);
        hipMemcpy(result.mutable_data(), d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);
        
        // Free memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        
        return result;
    }, "Perform vector addition on GPU");

    m.def("relu", [](py::array_t<float> input) {
        size_t numElements = input.size();
        
        // Allocate device memory
        float *d_input, *d_output;
        hipMalloc(&d_input, numElements * sizeof(float));
        hipMalloc(&d_output, numElements * sizeof(float));
        
        // Copy data to device
        hipMemcpy(d_input, input.data(), numElements * sizeof(float), hipMemcpyHostToDevice);
        
        // Run kernel
        int threadsPerBlock = 256;
        reluCUDA(d_input, d_output, numElements, threadsPerBlock);
        
        // Copy result back
        py::array_t<float> result(numElements);
        hipMemcpy(result.mutable_data(), d_output, numElements * sizeof(float), hipMemcpyDeviceToHost);
        
        // Free memory
        hipFree(d_input);
        hipFree(d_output);
        
        return result;
    }, "Apply ReLU on GPU");
}

